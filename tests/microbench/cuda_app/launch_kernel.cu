#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char **argv)
{
    const int size = 1000000;
    const int iterations = 10000;
    int a[size] = { 0 };
    int *dev_a = nullptr;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void **)&dev_a, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    for (int i = 0; i < iterations; i++) {
        addKernel<<<2, (size + 1) / 2>>>(dev_a, dev_a, dev_a, size);
        hipDeviceSynchronize();
    }

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);

    return 0;
}
